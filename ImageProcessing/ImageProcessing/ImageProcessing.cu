#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "hip/hip_runtime.h"
#include ""
#include "stb_image.h"
#include "stb_image_write.h"
#include <iostream>

__global__ void generateSepiaImage(float *inputImage, float *outputImage, int nCols, int nRows, int nChannels) {
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int x = threadIdx.x + blockIdx.x * blockDim.x;

    if (x >= nCols || y >= nRows) return;

    int pixelIdx = (y * nCols + x) * nChannels;
    float pixelValueR = inputImage[pixelIdx];
    float pixelValueG = inputImage[pixelIdx + 1];
    float pixelValueB = inputImage[pixelIdx + 2];

    float outR = pixelValueR * 0.393f + pixelValueG * 0.769f + pixelValueB * 0.189f;
    float outG = pixelValueR * 0.349f + pixelValueG * 0.686f + pixelValueB * 0.168f;
    float outB = pixelValueR * 0.272f + pixelValueG * 0.534f + pixelValueB * 0.131f;

    outputImage[pixelIdx] = fminf(fmaxf(outR, 0.0f), 255.0f);
    outputImage[pixelIdx + 1] = fminf(fmaxf(outG, 0.0f), 255.0f);
    outputImage[pixelIdx + 2] = fminf(fmaxf(outB, 0.0f), 255.0f);
}

__global__ void generateGrayscaleImage(float *inputImage, float *outputImage, int nCols, int nRows, int nChannels) {
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int x = threadIdx.x + blockIdx.x * blockDim.x;

    if (x >= nCols || y >= nRows) return;

    int pixelIdx = (y * nCols + x) * nChannels;
    float pixelValueR = inputImage[pixelIdx];
    float pixelValueG = inputImage[pixelIdx + 1];
    float pixelValueB = inputImage[pixelIdx + 2];

    float grayscaleValue = (pixelValueR + pixelValueG + pixelValueB) / 3.0;

    outputImage[pixelIdx] = grayscaleValue;
    outputImage[pixelIdx + 1] = grayscaleValue;
    outputImage[pixelIdx + 2] = grayscaleValue;
}

__global__ void horizontalBlur(float *inputImage, float *outputImage, int nRows, int nCols, int nChannels, int blurRadius) {
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int x = threadIdx.x + blockIdx.x * blockDim.x;

    if (x >= nCols || y >= nRows) return;

    float rSum = 0.0f, gSum = 0.0f, bSum = 0.0f;
    int totalValues = 0;

    for (int i = -blurRadius; i <= blurRadius; i++) {
        int xi = x + i;
        if (xi >= 0 && xi < nCols) {
            int neighborIdx = (y * nCols + xi) * nChannels;
            rSum += inputImage[neighborIdx];
            gSum += inputImage[neighborIdx + 1];
            bSum += inputImage[neighborIdx + 2];
            totalValues++;
        }
    }

    int pixelIdx = (y * nCols + x) * nChannels;
    outputImage[pixelIdx] = rSum / totalValues;
    outputImage[pixelIdx + 1] = gSum / totalValues;
    outputImage[pixelIdx + 2] = bSum / totalValues;
}

__global__ void verticalBlur(float *inputImage, float *outputImage, int nRows, int nCols, int nChannels, int blurRadius) {
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int x = threadIdx.x + blockIdx.x * blockDim.x;

    if (x >= nCols || y >= nRows) return;

    float rSum = 0.0f, gSum = 0.0f, bSum = 0.0f;
    int totalValues = 0;

    for (int i = -blurRadius; i <= blurRadius; i++) {
        int yi = y + i;
        if (yi >= 0 && yi < nRows) {
            int neighborIdx = (yi * nCols + x) * nChannels;
            rSum += inputImage[neighborIdx];
            gSum += inputImage[neighborIdx + 1];
            bSum += inputImage[neighborIdx + 2];
            totalValues++;
        }
    }

    int pixelIdx = (y * nCols + x) * nChannels;
    outputImage[pixelIdx] = rSum / totalValues;
    outputImage[pixelIdx + 1] = gSum / totalValues;
    outputImage[pixelIdx + 2] = bSum / totalValues;
}

bool checkInputArgs(int argc, char *argv[]) {
    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " <filter: sepia|blur> <image_file>" << std::endl;
        return false;
    }
    return true;
}

bool checkImage(unsigned char* imgDataChar, int nChannels) {
    if (imgDataChar == nullptr) {
        std::cerr << "Error loading image: " << stbi_failure_reason() << std::endl;
        return false;
    }
    else if (nChannels != 3 && nChannels != 4) {
        std::cout << "Sepia filter requires 3 or 4 channels, this image only has " << nChannels << " channels" << std::endl;
        return false;
    }
    return true;
}

int main(int argc, char *argv[]) {
    if (!checkInputArgs(argc, argv)) {return 1;}

    std::string filter = argv[1];
    const char* imagePath = argv[2];

    // Load the image
    int width, height, nChannels;
    unsigned char* imgDataChar = stbi_load(imagePath, &width, &height, &nChannels, 0);
    if (!checkImage(imgDataChar, nChannels)) {return 1;}
    
    
    // Calculate the number of elements 
    size_t nElements = width * height * nChannels;

    // Allocate memory for the image data in int format
    float *hImgData = new float[nElements];

    // Convert the data from char to float
    for (size_t i = 0; i < nElements; i++) {
        hImgData[i] = static_cast<float>(imgDataChar[i]);
    }
    

    float *dImgDataIn, *dImgDataOut;
    size_t nBytes = nElements * sizeof(float);
    hipMalloc((void **)&dImgDataIn, nBytes);
    hipMalloc((void **)&dImgDataOut, nBytes);
    hipMemcpy(dImgDataIn, hImgData, nBytes, hipMemcpyHostToDevice);

    dim3 block(16, 16, 1);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    // Call the appropriate kernel
    if (filter == "sepia") {
        generateSepiaImage<<<grid, block>>>(dImgDataIn, dImgDataOut, width, height, nChannels);
    } else if (filter == "hblur") {
        int blurLength = 10;
        horizontalBlur<<<grid, block>>>(dImgDataIn, dImgDataOut, width, height, nChannels, blurLength);
    } else if (filter == "vblur") {
        int blurLength = 10;
        verticalBlur<<<grid, block>>>(dImgDataIn, dImgDataOut, width, height, nChannels, blurLength);
    } else if (filter == "grayscale") {
        generateGrayscaleImage<<<grid, block>>>(dImgDataIn, dImgDataOut, width, height, nChannels);
    } else {
        std::cerr << "Unknown filter: " << filter << ". Use 'sepia' or 'blur'." << std::endl;
        return 1;
    }

    hipDeviceSynchronize();
    hipMemcpy(hImgData, dImgDataOut, nBytes, hipMemcpyDeviceToHost);


    for (size_t i = 0; i < nElements; i++) {
        imgDataChar[i] = static_cast<unsigned char>(hImgData[i]); 
    }

    // Save with different filename depending on filter
    std::string outputFilename = filter + "Image.png";
    if (!stbi_write_png(outputFilename.c_str(), width, height, nChannels, imgDataChar, width * nChannels)) {
        std::cerr << "Error saving image!" << std::endl;
    }

    
    stbi_image_free(imgDataChar);
    delete [] hImgData;
    hipFree(dImgDataIn);
    hipFree(dImgDataOut);
    hipDeviceReset();

    return 0;
}